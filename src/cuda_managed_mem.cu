#include "hip/hip_runtime.h"
/*

The MIT License (MIT)

Copyright (c) 2021 NVIDIA CORPORATION

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

*/


/*----------------------------------------------------------------------------
 * Standard C/C++ library headers
 *----------------------------------------------------------------------------*/
#include <assert.h>
#include <stdio.h>

#include <algorithm>
#include <list>

#include <cstdio>

/*----------------------------------------------------------------------------
 *  Local headers
 *----------------------------------------------------------------------------*/
#include "cuda_managed_mem.h"

#ifndef LOGGING
#define LOGGING 0
#endif

struct kv
{
    void* ptr;
    size_t size;
};

size_t sg_malloc_entry_count = 0 ;
kv* sg_malloc_entries = 0 ;

FILE* sg_log = 0 ;

int sg_malloc_has(void* ptr)
{
    for (size_t k = 0 ; k < sg_malloc_entry_count ; ++k)
    {
        if (sg_malloc_entries[k].ptr == ptr)
        {
            return 1;
        }
    }
    return 0;
}

void sg_malloc_put(void* ptr, size_t sz)
{
    #if LOGGING
    if (sg_log == 0) sg_log = fopen("nvidia-malloc.log", "w");
    fprintf(sg_log, "sg_malloc_put %p = %zu\n", ptr, sz);
    fclose(sg_log);
    #endif
    for (size_t k = 0 ; k < sg_malloc_entry_count ; ++k)
    {
        if (sg_malloc_entries[k].ptr == ptr)
        {
            sg_malloc_entries[k].size = sz;
            return;
        }
    }
    sg_malloc_entries = (kv*)realloc(sg_malloc_entries, (sg_malloc_entry_count+1)*sizeof(kv));
    sg_malloc_entries[sg_malloc_entry_count].ptr = ptr;
    sg_malloc_entries[sg_malloc_entry_count].size = sz;
    ++sg_malloc_entry_count;
}

size_t sg_malloc_get(void* ptr)
{
    #if LOGGING
    if (sg_log == 0) sg_log = fopen("nvidia-malloc.log", "w");
    fprintf(sg_log, "sg_malloc_get %p \n", ptr);
    fclose(sg_log);
    #endif
    for (size_t k = 0 ; k < sg_malloc_entry_count ; ++k)
    {
        if (sg_malloc_entries[k].ptr == ptr)
        {
            fprintf(sg_log, "\treturning sg_malloc_get %p = %zu \n", ptr, sg_malloc_entries[k].size);
            #if LOGGING
            fclose(sg_log);
            #endif
            return sg_malloc_entries[k].size;
        }
    }
    #if LOGGING
    fprintf(sg_log, "sg_malloc_get %p == NOT FOUND ==\n", ptr);
    fclose(sg_log);
    #endif
    return (size_t)0;
}

void sg_malloc_erase(void* ptr)
{
    #if LOGGING
    if (sg_log == 0) sg_log = fopen("nvidia-malloc.log", "w");
    fprintf(sg_log, "sg_malloc_erase %p \n", ptr);
    fclose(sg_log);
    #endif
    for (size_t k = 0 ; k < sg_malloc_entry_count ; ++k)
    {
        if (sg_malloc_entries[k].ptr == ptr)
        {
            // found !
            for (size_t j = k ; j < sg_malloc_entry_count-1 ; ++j)
            {
                sg_malloc_entries[j].ptr = sg_malloc_entries[j+1].ptr;
                sg_malloc_entries[j].size = sg_malloc_entries[j+1].size;
            }
            sg_malloc_entries = (kv*)realloc(sg_malloc_entries, (sg_malloc_entry_count-1)*sizeof(kv));
            --sg_malloc_entry_count;
            return;
        }
    }
}

#ifndef USEREALLOC
#define USEREALLOC 0
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {

#define CUDA_CHECK_LINE(a,file,line) {                                          \
    hipError_t __cuer = a;                                                     \
    if (hipSuccess != __cuer) {                                                \
        ::fprintf (stderr, "[CUDA-ERRROR] @ %s:%d -- %d : %s -- running %s\n",  \
                file,line, __cuer, ::hipGetErrorString(__cuer),#a) ;           \
        ::exit(__cuer) ;                                                        \
    }                                                                           \
}

#define CUDA_CHECK(a) CUDA_CHECK_LINE(a,__FILE__,__LINE__)

void cuda_managed_mem_malloc(void **pointer, size_t size)
{
    #if USEREALLOC
    *pointer = malloc(size);
    #else
    CUDA_CHECK(hipMallocManaged (pointer, size, hipMemAttachGlobal));
    // START ON HOST (seems that getcwd fails otherwise - System error: Bad address)
    CUDA_CHECK(hipMemPrefetchAsync (*pointer, size, hipCpuDeviceId, 0));
    CUDA_CHECK(hipDeviceSynchronize());

    sg_malloc_put(*pointer, size);
    #endif
} 

void cuda_managed_mem_free(void *pointer){
    #if USEREALLOC
    free(pointer);
    #else
    CUDA_CHECK(hipFree(pointer));
    CUDA_CHECK(hipDeviceSynchronize());
    sg_malloc_erase(pointer);
    #endif
}

void* cuda_managed_mem_realloc(void *pointer, size_t size)
{
    #if USEREALLOC
    return realloc(pointer, size);
    #else
    void* res;
    // http://www.cplusplus.com/reference/cstdlib/realloc/
    if (pointer != 0)
    {
        size_t prevsz ;
        // has it been previously freed ?
        if (sg_malloc_has(pointer) == 0)
        {
            // pointer has been freed, yet used again...
            #if LOGGING
            if (sg_log == 0) sg_log = fopen("nvidia-malloc.log", "w");
            fprintf(sg_log, "WARNING !! realloc with an unknown (non null) pointer = %p - no memcpy will be performed !\n", pointer);
            fclose(sg_log);
            #endif
            prevsz = 0;
        } else 
            prevsz = sg_malloc_get(pointer);
        if (prevsz != 0)
        {
            CUDA_CHECK(hipMemPrefetchAsync (pointer, prevsz, hipCpuDeviceId, 0));
            CUDA_CHECK(hipDeviceSynchronize());
        }
        cs_cuda_mem_malloc(&res, size);
        size_t minsz = prevsz < size ? prevsz : size;
        memcpy(res, pointer, minsz);
        // CUDA_CHECK(hipMemcpy(res, pointer, minsz, hipMemcpyDefault));
        CUDA_CHECK(hipDeviceSynchronize());
        cs_cuda_mem_free(pointer);
    } else 
        cs_cuda_mem_malloc(&res, size);
    return res;
    #endif
}

}
